
#include <hip/hip_runtime.h>
#include <cstdio>

#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void kernel(int a) {
  (void)a;
  __syncthreads();
}

int main(int argc, char **argv) {
  int deviceCount;

  gpuErrchk(hipGetDeviceCount(&deviceCount));
  printf("Detected %d devices\n", deviceCount);

  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, dev);

    if (dev == 0) {
      if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
        printf("No CUDA GPU has been detected.\n");
        return EXIT_FAILURE;
      } else {
        printf("There are %d device(s) supporting CUDA\n", deviceCount);
      }
    }

    printf("Device %d name: %s\n", dev, deviceProp.name);
    printf("  Computational Capabilities: %d.%d\n", deviceProp.major,
           deviceProp.minor);
    printf("  Maximum global memory size: %lu\n", deviceProp.totalGlobalMem);
    printf("  Maximum constant memory size: %lu\n", deviceProp.totalConstMem);
    printf("  Maximum shared memory size per block: %lu\n",
           deviceProp.sharedMemPerBlock);
    printf("  Maximum block dimensions: %d x %d x %d\n",
           deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1],
           deviceProp.maxThreadsDim[2]);
    printf("  Maximum grid dimensions: %d x %d x %d\n",
           deviceProp.maxGridSize[0], deviceProp.maxGridSize[1],
           deviceProp.maxGridSize[2]);
    printf("  Warp size: %d\n", deviceProp.warpSize);

    hipSetDevice(dev);
    printf("  Launching test kernel...");
    kernel<<<1, 1>>>(0); // test kernel launch
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    printf("Success!\n");
  }

  return 0;
}
